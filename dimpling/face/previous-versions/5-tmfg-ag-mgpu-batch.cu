#include <algorithm>
#include <cmath>
#include <cstdio>
#include <cstdlib>
#include <hip/hip_runtime.h>
#include <iomanip>
#include <iostream>
#include <omp.h>
#include <vector>

#define C 4
#define THREADS 1024 // 2^10
#define MAX 110
#define MAX_S MAX* MAX
#define PERM_MAX (MAX * (MAX - 1) * (MAX - 2) * (MAX - 3)) / 24
#define pb push_back
#define mp make_pair

#define gpuErrChk(ans)                        \
    {                                         \
        gpuAssert((ans), __FILE__, __LINE__); \
    }
inline void gpuAssert(hipError_t code, char* file, int line,
    bool abort = true)
{
    if (code != hipSuccess) {
        fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code),
            file, line);
        if (abort)
            getchar();
    }
}

using namespace std;

typedef long long int64;
typedef pair<int, int> ii;

/*
    sz          ---> Adjacency matrix dimension (1D)
    perm        ---> Number of permutations of an instance
    graph       ---> Adjacency matrix itself
    seeds       ---> Set of seeds
    faces       ---> Set of triangular faces for the output
    */
struct Node {
    int sz, perm;
    int graph[MAX_S], seeds[C * PERM_MAX], F_ANS[6 * MAX];
};

/*
    faces       ---> Number of triangular faces
    count       ---> Number of remaining vertices
    tmpMax      ---> Max value obtained for a seed
    F           ---> Set of triangular faces
    V           ---> Set of remaining vertices
    */
struct Params {
    int *faces, *count, *tmpMax;
    int *F, *V;
};

/*
    SIZE        ---> Number of vertices
    BLOCKS      ---> Number of blocks
    PERM        ---> Number of permutations
    R           ---> Output graph for a possible solution
    F           ---> Set of triangular faces of an instance
    qtd         ---> Number of possible 4-cliques
    */
int SIZE, PERM, GPU_CNT = 1;
int R[MAX_S], F[6 * MAX], bib[MAX];
int qtd = 0;

Node* N;

/*
    Generates a list containing the vertices which are not on the planar graph.
    */
__device__ void generateList(Node* devN, Params* devP, int t, int offset)
{
    int sz = devN->sz, perm = devN->perm;

    int va = devN->seeds[(t + offset) * 4],
        vb = devN->seeds[(t + offset) * 4 + 1],
        vc = devN->seeds[(t + offset) * 4 + 2],
        vd = devN->seeds[(t + offset) * 4 + 3];
    for (int i = 0; i < sz; i++) {
        if (i == va || i == vb || i == vc || i == vd)
            devP->V[t + i * perm] = -1;
        else
            devP->V[t + i * perm] = i;
    }
}

/*
    Returns the weight of the planar graph so far.
    */
__device__ void generateTriangularFaceList(Node* devN, Params* devP, int graph[],
    int t, int offset)
{
    int sz = devN->sz, perm = devN->perm;

    int va = devN->seeds[(t + offset) * 4],
        vb = devN->seeds[(t + offset) * 4 + 1],
        vc = devN->seeds[(t + offset) * 4 + 2],
        vd = devN->seeds[(t + offset) * 4 + 3];

    /* Generate first triangle of the output graph */
    devP->F[t + (devP->faces[t] * 3) * perm] = va;
    devP->F[t + (devP->faces[t] * 3 + 1) * perm] = vb;
    devP->F[t + ((devP->faces[t]++) * 3 + 2) * perm] = vc;

    /* Generate the next 3 possible faces */
    devP->F[t + (devP->faces[t] * 3) * perm] = va;
    devP->F[t + (devP->faces[t] * 3 + 1) * perm] = vb;
    devP->F[t + ((devP->faces[t]++) * 3 + 2) * perm] = vd;

    devP->F[t + (devP->faces[t] * 3) * perm] = va;
    devP->F[t + (devP->faces[t] * 3 + 1) * perm] = vc;
    devP->F[t + ((devP->faces[t]++) * 3 + 2) * perm] = vd;

    devP->F[t + (devP->faces[t] * 3) * perm] = vb;
    devP->F[t + (devP->faces[t] * 3 + 1) * perm] = vc;
    devP->F[t + ((devP->faces[t]++) * 3 + 2) * perm] = vd;

    int resp = graph[va * sz + vb] + graph[va * sz + vc] + graph[vb * sz + vc];
    resp += graph[va * sz + vd] + graph[vb * sz + vd] + graph[vc * sz + vd];

    devP->tmpMax[t] = resp;
}

/*
    Insert a new vertex, 3 new triangular faces and removes face 'f'
    from the set.
    */
__device__ int operationT2(Node* devN, Params* devP, int graph[],
    int new_vertex, int f, int t)
{
    int sz = devN->sz, perm = devN->perm;

    /* Remove the chosen face and insert a new one */
    int va = devP->F[t + (f * 3) * perm],
        vb = devP->F[t + (f * 3 + 1) * perm],
        vc = devP->F[t + (f * 3 + 2) * perm];

    devP->F[t + (f * 3) * perm] = new_vertex,
                          devP->F[t + (f * 3 + 1) * perm] = va,
                          devP->F[t + (f * 3 + 2) * perm] = vb;

    /* and insert the other two possible faces. */
    devP->F[t + (devP->faces[t] * 3) * perm] = new_vertex;
    devP->F[t + (devP->faces[t] * 3 + 1) * perm] = va;
    devP->F[t + ((devP->faces[t]++) * 3 + 2) * perm] = vc;

    devP->F[t + (devP->faces[t] * 3) * perm] = new_vertex;
    devP->F[t + (devP->faces[t] * 3 + 1) * perm] = vb;
    devP->F[t + ((devP->faces[t]++) * 3 + 2) * perm] = vc;

    int resp = graph[va * sz + new_vertex] + graph[vb * sz + new_vertex] + graph[vc * sz + new_vertex];

    return resp;
}

/*
    Return the vertex with the maximum gain inserting within a face 'f'.
    */
__device__ int maxGain(Node* devN, Params* devP, int graph[], int* f, int t)
{
    int sz = devN->sz, perm = devN->perm;
    int gain = -1, vertex = -1;

    /* iterate through the remaining vertices */
    for (int new_vertex = 0; new_vertex < sz; new_vertex++) {
        if (devP->V[t + new_vertex * perm] == -1)
            continue;
        /* and test which has the maximum gain with its insetion
            within all possible faces */
        int faces = devP->faces[t];
        for (int i = 0; i < faces; i++) {
            int va = devP->F[t + (i * 3) * perm],
                vb = devP->F[t + (i * 3 + 1) * perm],
                vc = devP->F[t + (i * 3 + 2) * perm];
            int tmpGain = graph[va * sz + new_vertex] + graph[vb * sz + new_vertex]
                + graph[vc * sz + new_vertex];
            if (tmpGain > gain) {
                gain = tmpGain;
                *f = i;
                vertex = new_vertex;
            }
        }
    }
    return vertex;
}

__device__ void tmfg(Node* devN, Params* devP, int graph[], int t)
{
    int perm = devN->perm;
    while (devP->count[t]) {
        int f = -1;
        int vertex = maxGain(devN, devP, graph, &f, t);
        devP->V[t + vertex * perm] = -1;
        devP->tmpMax[t] += operationT2(devN, devP, graph, vertex, f, t);
        devP->count[t]--;
    }
}

__device__ void copyGraph(Node* devN, Params* devP, int t)
{
    int faces = devP->faces[t], perm = devN->perm;
    for (int i = 0; i < faces; i++) {
        int va = devP->F[t + (i * 3) * perm],
            vb = devP->F[t + (i * 3 + 1) * perm],
            vc = devP->F[t + (i * 3 + 2) * perm];
        devN->F_ANS[i * 3] = va, devN->F_ANS[i * 3 + 1] = vb,
                        devN->F_ANS[i * 3 + 2] = vc;
    }
}

__device__ void initializeDevice(Params* devP, int sz, int t)
{
    devP->faces[t] = 0;
    devP->tmpMax[t] = -1;
    devP->count[t] = sz - 4;
}

__global__ void tmfgParallel(Node* devN, Params devP, int* respMax,
    int offset, int mx)
{
    int x = blockDim.x * blockIdx.x + threadIdx.x;
    devN->perm = mx;
    int sz = devN->sz, perm = devN->perm;
    /* Uncoment the following line to put the graph on the shared memory */
    // extern __shared__ int graph[];
    int* graph;

    /* Uncoment the following line to put the graph on the shared memory */
    // for (int i = threadIdx.x; i < sz*sz; i += blockDim.x)
    //     graph[i] = devN->graph[i];
    // __syncthreads();
    graph = devN->graph;

    if (x < mx && x < perm) {
        initializeDevice(&devP, sz, x);
        generateList(devN, &devP, x, offset);
        generateTriangularFaceList(devN, &devP, graph, x, offset);
        tmfg(devN, &devP, graph, x);
        atomicMax(respMax, devP.tmpMax[x]);
        __syncthreads();

        if (devP.tmpMax[x] == *respMax) {
            copyGraph(devN, &devP, x);
        }
        __syncthreads();
    }
}

int tmfgPrepare()
{
    int finalResp = -1, pos = -1;

#pragma omp parallel for num_threads(GPU_CNT)
    for (int gpu_id = 0; gpu_id < GPU_CNT; gpu_id++) {
        hipSetDevice(gpu_id);
        int range = (int)ceil(PERM / (double)GPU_CNT);
        int perm = ((gpu_id + 1) * range > PERM ? PERM - gpu_id * range : range);
        int offset = gpu_id * range;
        N->perm = perm;

        int resp = -1, *tmpResp;
        gpuErrChk(hipMalloc((void**)&tmpResp, sizeof(int)));
        gpuErrChk(hipMemcpy(tmpResp, &resp, sizeof(int), hipMemcpyHostToDevice));

        Node* devN;
        Params devP;

        gpuErrChk(hipMalloc((void**)&devN, sizeof(Node)));
        gpuErrChk(hipMemcpy(devN, N, sizeof(Node), hipMemcpyHostToDevice));

        size_t sz_node = sizeof(int) * MAX_S + sizeof(int) * C * PERM_MAX + sizeof(int) * 6 * MAX;
        size_t sz_prm = range * sizeof(int) * 3 + range * sizeof(int) * (7 * SIZE);

        printf("Using %d mbytes in Kernel %d\n", (sz_node + sz_prm) / (1 << 20), gpu_id);
        fprintf(stderr, "Using %d mbytes in Kernel %d\n", (sz_node + sz_prm) / (1 << 20), gpu_id);

        size_t cuInfo = 0, cuTotal = 0;
        gpuErrChk(hipMemGetInfo(&cuInfo, &cuTotal));
        cuInfo *= 0.95;
        printf("Free memory: %dMB\nTotal memory: %dMB\n", cuInfo / (1 << 20), cuTotal / (1 << 20));

        int it_range, it_perm, it_offset;
        int BATCH_CNT = (int)ceil(sz_prm / (double)cuInfo);
        printf("Iterations: %d\n", BATCH_CNT);
        it_range = (int)ceil(perm / (double)BATCH_CNT);

        gpuErrChk(hipMalloc((void**)&devP.faces, it_range * sizeof(int)));
        gpuErrChk(hipMalloc((void**)&devP.count, it_range * sizeof(int)));
        gpuErrChk(hipMalloc((void**)&devP.tmpMax, it_range * sizeof(int)));
        gpuErrChk(hipMalloc((void**)&devP.F, 6 * SIZE * it_range * sizeof(int)));
        gpuErrChk(hipMalloc((void**)&devP.V, SIZE * it_range * sizeof(int)));

        for (int btch_id = 0; btch_id < BATCH_CNT; btch_id++) {
            it_perm = ((btch_id + 1) * it_range > perm ? perm - btch_id * it_range : it_range);
            it_offset = btch_id * it_range + offset;

            dim3 blocks(it_perm / THREADS + 1, 1);
            dim3 threads(THREADS, 1);

            printf("Kernel %d launched with %d blocks, each w/ %d threads\n", btch_id + 1,
                it_range / THREADS + 1, THREADS);
            fprintf(stderr, "Kernel %d launched with %d blocks, each w/ %d threads\n",
                btch_id, it_range / THREADS + 1, THREADS);
            /* Uncoment the following line to put the graph on the shared memory */
            // tmfgParallel <<<blocks, threads, SIZE*SIZE*sizeof(int)>>>(devN, devP,
            // tmpResp, it_offset, it_perm);
            tmfgParallel<<<blocks, threads>>>(devN, devP, tmpResp, it_offset, it_perm);
            gpuErrChk(hipDeviceSynchronize());

            /*
                Copy back the maximum weight and the set of faces
                which gave this result.
                */
            gpuErrChk(hipMemcpy(&resp, tmpResp, sizeof(int), hipMemcpyDeviceToHost));
            printf("Kernel finished.\nLocal maximum found in Kernel %d: %d\n", btch_id, resp);

            printf("Copying results...\n");
#pragma omp critical
            {
                if (resp > finalResp) {
                    finalResp = resp;
                    pos = gpu_id;
                }
            }

            if (pos == gpu_id) {
                gpuErrChk(hipMemcpy(&F, devN->F_ANS, 6 * MAX * sizeof(int),
                    hipMemcpyDeviceToHost));
            }
        }

        printf("Freeing memory...\n");
        gpuErrChk(hipFree(devN));
        gpuErrChk(hipFree(devP.faces));
        gpuErrChk(hipFree(devP.count));
        gpuErrChk(hipFree(devP.tmpMax));
        gpuErrChk(hipFree(devP.F));
        gpuErrChk(hipFree(devP.V));

        gpuErrChk(hipDeviceReset());
    }

    return finalResp;
}

/*
    Print elapsed time.
    */
void printElapsedTime(double start, double stop)
{
    double elapsed = stop - start;
    printf("Elapsed time: %.3lfs.\n", elapsed);
}

double getTime()
{
    timespec ts;
    clock_gettime(CLOCK_REALTIME, &ts);
    return double(ts.tv_sec) + double(ts.tv_nsec) / 1e9;
}

/*
    C           ---> Size of the combination
    index       ---> Current index in data[]
    data[]      ---> Temporary array to store a current combination
    i           ---> Index of current element in vertices[]
    */
void combineUntil(int index, vector<int>& data, int i)
{
    // Current cobination is ready, print it
    if (index == C) {
        for (int j = 0; j < C; j++) {
            N->seeds[qtd * C + j] = data[j];
        }
        qtd++;
        return;
    }

    // When there are no more elements to put in data[]
    if (i >= SIZE)
        return;

    //current is inserted; put next at a next location
    data[index] = i;
    combineUntil(index + 1, data, i + 1);

    //current is deleted; replace it with next
    combineUntil(index, data, i + 1);
}

void combine()
{
    vector<int> data(C);
    /*
        print all combinations of size 'r' using a temporary array 'data'
        */
    combineUntil(0, data, 0);
}

void initialize()
{
    for (int i = 0; i < SIZE - 1; i++) {
        for (int j = i + 1; j < SIZE; j++) {
            R[i * SIZE + j] = R[j * SIZE + i] = -1;
        }
    }
}

void readInput()
{
    int x;
    cin >> SIZE;
    PERM = bib[SIZE - 1];

    N = (Node*)malloc(sizeof(Node));
    N->sz = SIZE;

    for (int i = 0; i < SIZE - 1; i++) {
        for (int j = i + 1; j < SIZE; j++) {
            cin >> x;
            N->graph[i * SIZE + j] = x;
            N->graph[j * SIZE + i] = x;
        }
    }
}

/*
    Define the number of permutations and blocks
    */
void sizeDefinitions()
{
    for (int i = 4; i <= MAX; i++) {
        int resp = 1;
        for (int j = i - 3; j <= i; j++)
            resp *= j;
        resp /= 24;
        bib[i - 1] = resp;
    }
}

int main(int argv, char** argc)
{
    ios::sync_with_stdio(false);
    sizeDefinitions();
    /*
        Read the input, which is given by a size of a graph and its weighted
        edges. The given graph is dense.
        */
    readInput();
    initialize();
    /*
        Given the number of vertices, generate multiple 4-clique seeds.
        */
    combine();

    if (argv == 2) {
        hipSetDevice(atoi(argc[1]));
    } else if (argv == 3) {
        GPU_CNT = atoi(argc[2]);
        int d;
        hipGetDeviceCount(&d);
        if (GPU_CNT > d)
            GPU_CNT = d;
    }

    double start = getTime();
    int respMax = tmfgPrepare();
    double stop = getTime();

    /*
        Reconstruct the graph given the faces of the graph
        */
    for (int i = 0; i < 2 * SIZE; i++) {
        int va = F[i * 3], vb = F[i * 3 + 1], vc = F[i * 3 + 2];
        if (va == vb && vb == vc)
            continue;
        R[va * SIZE + vb] = R[vb * SIZE + va] = N->graph[va * SIZE + vb];
        R[va * SIZE + vc] = R[vc * SIZE + va] = N->graph[va * SIZE + vc];
        R[vb * SIZE + vc] = R[vc * SIZE + vb] = N->graph[vb * SIZE + vc];
    }

    cout << "Printing generated graph: " << endl;
    for (int i = 0; i < SIZE - 1; i++) {
        for (int j = i + 1; j < SIZE; j++) {
            cout << R[i * SIZE + j] << " ";
        }
        cout << endl;
    }
    cout << endl;

    printElapsedTime(start, stop);
    cout << "Maximum weight found: " << respMax << endl;
    free(N);

    return 0;
}