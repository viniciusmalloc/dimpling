#include "hip/hip_runtime.h"
#include <algorithm>
#include <cmath>
#include <cstdio>
#include <cstdlib>
#include <hip/hip_runtime.h>
#include <iomanip>
#include <iostream>
#include <omp.h>
#include <vector>

#include "combinadic.h"
#include "default.h"

#define pb push_back
#define mp make_pair

#define gpuErrChk(ans)                        \
    {                                         \
        gpuAssert((ans), __FILE__, __LINE__); \
    }
inline void gpuAssert(hipError_t code, char* file, int line,
    bool abort = true)
{
    if (code != hipSuccess) {
        fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code),
            file, line);
        if (abort)
            getchar();
    }
}

using namespace std;

typedef pair<int, int> ii;
typedef unsigned long long uint64;

#define EPS 1e-9

__device__ int SGN(double a)
{
    return ((a > EPS) ? (1) : ((a < -EPS) ? (-1) : (0)));
}
__device__ int CMP(double a, double b) { return SGN(a - b); }

int hostSGN(double a) { return ((a > EPS) ? (1) : ((a < -EPS) ? (-1) : (0))); }
int hostCMP(double a, double b) { return hostSGN(a - b); }

__device__ void AtomicMax(double* const address, const double value)
{
    if (*address >= value)
        return;

    uint64* const address_as_i = (uint64*)address;
    uint64 old = *address_as_i, assumed;

    do {
        assumed = old;
        if (__longlong_as_double(assumed) >= value)
            break;
        old = atomicCAS(address_as_i, assumed, __double_as_longlong(value));
    } while (assumed != old);
}

/*  Shared Combinadic instance on the GPU
    */
__shared__ Combination c;

__device__ void generateList(Node* devN, Params* devP, int t, int offset)
{
    int sz = devN->sz;
    int perm = devN->perm;

    int* seeds = c.element(t + offset).getArray();

    int va = seeds[0], vb = seeds[1], vc = seeds[2], vd = seeds[3];
    for (int i = 0, pos = 0; i < sz; i++) {
        if (i != va && i != vb && i != vc && i != vd)
            devP->V[t + (pos++) * perm] = i;
    }
}

/*  Returns the weight of the initial planar subgraph.
    */
__device__ void generateTriangularFaceList(Node* devN, Params* devP, double graph[],
    int t, int offset)
{
    int sz = devN->sz;
    int perm = devN->perm;

    int* seeds = c.element(t + offset).getArray();

    int va = seeds[0], vb = seeds[1], vc = seeds[2], vd = seeds[3];

    /*  Generate first triangle of the output graph
        */
    devP->F[t + (devP->faces[t] * 3) * perm] = va;
    devP->F[t + (devP->faces[t] * 3 + 1) * perm] = vb;
    devP->F[t + ((devP->faces[t]++) * 3 + 2) * perm] = vc;

    /*  Generate the next 3 possible faces
        */
    devP->F[t + (devP->faces[t] * 3) * perm] = va;
    devP->F[t + (devP->faces[t] * 3 + 1) * perm] = vb;
    devP->F[t + ((devP->faces[t]++) * 3 + 2) * perm] = vd;

    devP->F[t + (devP->faces[t] * 3) * perm] = va;
    devP->F[t + (devP->faces[t] * 3 + 1) * perm] = vc;
    devP->F[t + ((devP->faces[t]++) * 3 + 2) * perm] = vd;

    devP->F[t + (devP->faces[t] * 3) * perm] = vb;
    devP->F[t + (devP->faces[t] * 3 + 1) * perm] = vc;
    devP->F[t + ((devP->faces[t]++) * 3 + 2) * perm] = vd;

    double resp = graph[va + sz * vb] + graph[va + sz * vc] + graph[vb + sz * vc];
    resp += graph[va + sz * vd] + graph[vb + sz * vd] + graph[vc + sz * vd];

    devP->tmpMax[t] = resp;
}

/*  Insert a new vertex, 3 new triangular faces and removes face 'f'
    from the set.
    */
__device__ double operationT2(Node* devN, Params* devP, double graph[],
    int new_vertex, int f, int t)
{
    int sz = devN->sz;
    int perm = devN->perm;

    /*  Remove the chosen face and insert a new one
        */
    int va = devP->F[t + (f * 3) * perm],
        vb = devP->F[t + (f * 3 + 1) * perm],
        vc = devP->F[t + (f * 3 + 2) * perm];

    devP->F[t + (f * 3) * perm] = new_vertex,
                          devP->F[t + (f * 3 + 1) * perm] = va,
                          devP->F[t + (f * 3 + 2) * perm] = vb;

    /*  and insert the other two possible faces.
        */
    devP->F[t + (devP->faces[t] * 3) * perm] = new_vertex;
    devP->F[t + (devP->faces[t] * 3 + 1) * perm] = va;
    devP->F[t + ((devP->faces[t]++) * 3 + 2) * perm] = vc;

    devP->F[t + (devP->faces[t] * 3) * perm] = new_vertex;
    devP->F[t + (devP->faces[t] * 3 + 1) * perm] = vb;
    devP->F[t + ((devP->faces[t]++) * 3 + 2) * perm] = vc;

    double resp = graph[va + sz * new_vertex] + graph[vb + sz * new_vertex] + graph[vc + sz * new_vertex];

    return resp;
}

/*  Return the vertex with the maximum gain inserting within a face 'f'.
    */
__device__ int maxGain(Node* devN, Params* devP, double graph[], int* f, int t)
{
    int sz = devN->sz;
    int perm = devN->perm;
    double gain = -1.0;
    int vertex = -1;

    /*  Iterate through the remaining vertices
        */
    int remain = devP->count[t];
    for (int r = 0; r < remain; r++) {
        int new_vertex = devP->V[t + r * perm];
        // if (new_vertex == -1) continue;
        /*  and test which has the maximum gain with its insetion
            within all possible faces
            */
        int faces = devP->faces[t];
        for (int i = 0; i < faces; i++) {
            int va = devP->F[t + (i * 3) * perm],
                vb = devP->F[t + (i * 3 + 1) * perm],
                vc = devP->F[t + (i * 3 + 2) * perm];
            double tmpGain = graph[va + sz * new_vertex] + graph[vb + sz * new_vertex]
                + graph[vc + sz * new_vertex];
            if (CMP(tmpGain, gain) == 1) {
                gain = tmpGain;
                *f = i;
                vertex = r;
            }
        }
    }
    return vertex;
}

__device__ void dimpling(Node* devN, Params* devP, double graph[], int t)
{
    int perm = devN->perm;

    while (devP->count[t]) {
        int last = devP->count[t] - 1;
        int f = -1;
        int vertex_idx = maxGain(devN, devP, graph, &f, t);
        int vertex = devP->V[t + vertex_idx * perm];
        devP->tmpMax[t] += operationT2(devN, devP, graph, vertex, f, t);

        for (int i = vertex_idx; i <= last; i++)
            devP->V[t + i * perm] = devP->V[t + (i + 1) * perm];
        devP->count[t]--;
    }
}

__device__ void copyGraph(Node* devN, Params* devP, int t)
{
    int faces = devP->faces[t];
    int perm = devN->perm;
    for (int i = 0; i < faces; i++) {
        int va = devP->F[t + (i * 3) * perm],
            vb = devP->F[t + (i * 3 + 1) * perm],
            vc = devP->F[t + (i * 3 + 2) * perm];
        devN->F_ANS[i * 3] = va, devN->F_ANS[i * 3 + 1] = vb,
                        devN->F_ANS[i * 3 + 2] = vc;
    }
}

__device__ void initializeDevice(Params* devP, int sz, int t)
{
    devP->faces[t] = 0;
    devP->tmpMax[t] = 0.0;
    devP->count[t] = sz - 4;
}

__global__ void dimplingKernel(Node* __restrict__ devN, Params devP,
    double* respMax, int offset, int perm)
{
    devN->perm = perm;
    int x = blockDim.x * blockIdx.x + threadIdx.x;
    int sz = devN->sz;

    if (threadIdx.x == 0)
        c = Combination(sz, 4);
    double* graph = devN->graph;
    __syncthreads();

    if (x < perm) {
        initializeDevice(&devP, sz, x);
        generateList(devN, &devP, x, offset);
        generateTriangularFaceList(devN, &devP, graph, x, offset);
        dimpling(devN, &devP, graph, x);
        AtomicMax(respMax, devP.tmpMax[x]);
        __syncthreads();

        if (devP.tmpMax[x] == *respMax) {
            copyGraph(devN, &devP, x);
        }
        __syncthreads();
    }
}

__global__ void dimplingKernelShared(Node* __restrict__ devN, Params devP,
    double* respMax, int offset, int perm)
{
    devN->perm = perm;
    int x = blockDim.x * blockIdx.x + threadIdx.x;
    int sz = devN->sz;

    if (threadIdx.x == 0)
        c = Combination(sz, 4);
    extern __shared__ double graph[];
    for (int i = threadIdx.x; i < sz * sz; i += blockDim.x)
        graph[i] = devN->graph[i];
    __syncthreads();

    if (x < perm) {
        initializeDevice(&devP, sz, x);
        generateList(devN, &devP, x, offset);
        generateTriangularFaceList(devN, &devP, graph, x, offset);
        dimpling(devN, &devP, graph, x);
        AtomicMax(respMax, devP.tmpMax[x]);
        __syncthreads();

        if (devP.tmpMax[x] == *respMax) {
            copyGraph(devN, &devP, x);
        }
        __syncthreads();
    }
}

double dimplingPrepare(int sharedOn)
{
    double finalResp = 0.0;
    int pos = -1;

#pragma omp parallel num_threads(GPU_CNT)
    {
        int gpu_id = omp_get_thread_num();
        hipSetDevice(gpu_id);

        /*  range        ---> Range of the seeds in the GPU
            perm         ---> Number of seeds divided between the GPUs
            offset       ---> Offset for each GPU
            */
        int64 range = (int)ceil(PERM / (double)GPU_CNT);
        int64 perm = ((gpu_id + 1) * range > PERM ? PERM - gpu_id * range : range);
        int64 offset = gpu_id * range;

        /*  Create a temporary result variable on the GPU and set its value to -1.
            */
        double resp = 0.0, *tmpResp;
        gpuErrChk(hipMalloc((void**)&tmpResp, sizeof(double)));
        gpuErrChk(hipMemcpy(tmpResp, &resp, sizeof(double), hipMemcpyHostToDevice));

        Node* devN;
        Params devP;

        /*  Create an instance of Node on the GPU and copy the values on the host
            to it.
            */
        gpuErrChk(hipMalloc((void**)&devN, sizeof(Node)));
        gpuErrChk(hipMemcpy(devN, N, sizeof(Node), hipMemcpyHostToDevice));

        /*  Calculate the required amount of space to run the instance on the GPU.
            */
        size_t sz_node = sizeof(double) * MAX_S + sizeof(int) * 6 * MAX + sizeof(int) * 2;
        size_t sz_prm = range * sizeof(int) * 2 + range * sizeof(double) + range * sizeof(int) * (7 * SIZE);

        // printf("Using %d mbytes in Kernel %d\n", (sz_node + sz_prm) / (1 << 20), gpu_id);
        fprintf(stderr, "Using %d mbytes on Kernel %d\n", (sz_node + sz_prm) / (1 << 20), gpu_id);

        size_t cuInfo = 0, cuTotal = 0;
        gpuErrChk(hipMemGetInfo(&cuInfo, &cuTotal));
        cuInfo *= 0.95;
        printf("Free memory: %d mbytes\nTotal memory: %d mbytes\n", cuInfo / (1 << 20), cuTotal / (1 << 20));

        /*  BATCH_CNT       ---> Number of calls to the kernel for each GPU
            it_range        ---> Range of the seeds in the batch
            it_perm         ---> Number of seeds divided between the batches
            it_offset       ---> Offset for each batch
            */
        int BATCH_CNT = (int)ceil(sz_prm / (double)cuInfo);
        int it_range = (int)ceil(perm / (double)BATCH_CNT);
        int it_perm, it_offset;
        printf("Required num. of iterations: %d\n", BATCH_CNT);

        /*  Reserve the require amount of space for each variable on Params.
            */
        gpuErrChk(hipMalloc((void**)&devP.faces, it_range * sizeof(int)));
        gpuErrChk(hipMalloc((void**)&devP.count, it_range * sizeof(int)));
        gpuErrChk(hipMalloc((void**)&devP.tmpMax, it_range * sizeof(double)));
        gpuErrChk(hipMalloc((void**)&devP.F, 6 * SIZE * it_range * sizeof(int)));
        gpuErrChk(hipMalloc((void**)&devP.V, SIZE * it_range * sizeof(int)));

        // printf("Kernel %d launched with %lld blocks, each w/ %d threads\n", gpu_id+1,
        // it_range / THREADS+1, THREADS);
        fprintf(stderr, "Kernel %d launched with %d blocks, each w/ %d threads\n",
            gpu_id + 1, it_range / THREADS + 1, THREADS);

        for (int btch_id = 0; btch_id < BATCH_CNT; btch_id++) {
            it_perm = ((btch_id + 1) * it_range > perm ? perm - btch_id * it_range : it_range);
            it_offset = btch_id * it_range + offset;

            dim3 blocks(it_perm / THREADS + 1, 1);
            dim3 threads(THREADS, 1);

            if (SIZE > 75 || !sharedOn) {
                dimplingKernel<<<blocks, threads>>>(devN, devP, tmpResp, it_offset, it_perm);
                gpuErrChk(hipDeviceSynchronize());
            } else {
                dimplingKernelShared<<<blocks, threads, SIZE * SIZE * sizeof(double)>>>(devN, devP,
                    tmpResp, it_offset, it_perm);
                gpuErrChk(hipDeviceSynchronize());
            }

            /*  Copy the maximum weight found.
                */
            gpuErrChk(hipMemcpy(&resp, tmpResp, sizeof(double), hipMemcpyDeviceToHost));

            /*  The result obtained by each GPU will only be copied if its value is higher
                than the current one.
                */
            printf("Kernel finished with local maximum %.3lf. Copying results...\n", resp);

#pragma omp barrier
            {
#pragma omp critical
                {
                    if (hostCMP(resp, finalResp) == 1) {
                        finalResp = resp;
                        pos = gpu_id;
                    }
                }
            }

            if (pos == gpu_id) {
                gpuErrChk(hipMemcpy(&F, devN->F_ANS, 6 * MAX * sizeof(int),
                    hipMemcpyDeviceToHost));
            }
        }

        printf("Freeing memory...\n");
        gpuErrChk(hipFree(devN));
        gpuErrChk(hipFree(devP.faces));
        gpuErrChk(hipFree(devP.count));
        gpuErrChk(hipFree(devP.tmpMax));
        gpuErrChk(hipFree(devP.F));
        gpuErrChk(hipFree(devP.V));

        gpuErrChk(hipDeviceReset());
    }

    return finalResp;
}

int main(int argv, char** argc)
{
    sizeDefinitions();
    /*  Read the input, which is given by a size of a graph and its weighted
        edges. The given graph is dense.
        */
    readInput();
    initialize();

    int sharedOn = 0;
    if (argv == 2) {
        sharedOn = 1;
        hipSetDevice(atoi(argc[1]));
    } else if (argv == 3) {
        sharedOn = atoi(argc[1]);
        GPU_CNT = atoi(argc[2]);
        int d;
        hipGetDeviceCount(&d);
        if (GPU_CNT > d)
            GPU_CNT = d;
    } else {
        cout << "ERROR! Minimum num. of arguments: 1\n"
                "Try:\nsimple gpu - ./a.out gpu_id\nmulti-gpu  - ./a.out sharedOnOff num_gpus\n";
        return 0;
    }

    double start = getTime();
    double respMax = dimplingPrepare(sharedOn);
    double stop = getTime();

    /*  Reconstruct the graph given the faces of the graph
        */
    for (int i = 0; i < 2 * SIZE; i++) {
        int va = F[i * 3], vb = F[i * 3 + 1], vc = F[i * 3 + 2];
        //outbounds verification
        if (va == vb && vb == vc)
            continue;
        R[va * SIZE + vb] = R[vb * SIZE + va] = N->graph[va * SIZE + vb];
        R[va * SIZE + vc] = R[vc * SIZE + va] = N->graph[va * SIZE + vc];
        R[vb * SIZE + vc] = R[vc * SIZE + vb] = N->graph[vb * SIZE + vc];
    }

    cout << "Printing generated graph: " << endl;
    for (int i = 0; i < SIZE; i++) {
        for (int j = i + 1; j < SIZE; j++) {
            printf("%lf ", (R[i * SIZE + j] == -1 ? 0 : R[i * SIZE + j]));
        }
        printf("\n");
    }

    printElapsedTime(start, stop);
    printf("Maximum weight found: %.3lf\n", respMax);
    free(N);

    return 0;
}
